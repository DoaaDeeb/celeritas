#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file SeltzerBerger.cu
//---------------------------------------------------------------------------//
#include "SeltzerBerger.hh"

#include "base/Assert.hh"
#include "base/KernelParamCalculator.cuda.hh"
#include "random/cuda/RngEngine.hh"
#include "physics/base/ModelInterface.hh"
#include "physics/base/ParticleTrackView.hh"
#include "physics/base/PhysicsTrackView.hh"
#include "physics/base/SecondaryAllocatorView.hh"
#include "physics/material/MaterialTrackView.hh"
#include "SeltzerBergerInteractor.hh"

namespace celeritas
{
namespace detail
{
namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
/*!
 * Interact using the Seltzer-Berger model on applicable tracks.
 */
__global__ void seltzer_berger_interact_kernel(const SeltzerBergerPointers bh,
                                               const ModelInteractPointers ptrs)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= ptrs.states.size())
        return;

    SecondaryAllocatorView allocate_secondaries(ptrs.secondaries);
    ParticleTrackView particle(ptrs.params.particle, ptrs.states.particle, tid);

    // Setup for ElementView access
    MaterialTrackView material(ptrs.params.material, ptrs.states.material, tid);
    // Cache the associated MaterialView as function calls to MaterialTrackView
    // are expensive
    MaterialView material_view = material.material_view();

    PhysicsTrackView physics(ptrs.params.physics,
                             ptrs.states.physics,
                             particle.particle_id(),
                             material.material_id(),
                             tid);

    // This interaction only applies if the Seltzer-Berger model was selected
    if (physics.model_id() != bh.model_id)
        return;

    // Assume only a single element in the material, for now
    CELER_ASSERT(material_view.num_elements() == 1);
    SeltzerBergerInteractor interact(
        bh,
        particle,
        ptrs.states.direction[tid.get()],
        allocate_secondaries,
        material_view.element_view(celeritas::ElementComponentId{0}));

    RngEngine rng(ptrs.states.rng, tid);
    ptrs.result[tid.get()] = interact(rng);
    CELER_ENSURE(ptrs.result[tid.get()]);
}

} // namespace

//---------------------------------------------------------------------------//
// LAUNCHERS
//---------------------------------------------------------------------------//
/*!
 * Launch the Seltzer-Berger interaction.
 */
void seltzer_berger_interact(const SeltzerBergerPointers& bh,
                             const ModelInteractPointers& model)
{
    CELER_EXPECT(bh);
    CELER_EXPECT(model);

    static const KernelParamCalculator calc_kernel_params(
        seltzer_berger_interact_kernel, "seltzer_berger_interact");
    auto params = calc_kernel_params(model.states.size());
    seltzer_berger_interact_kernel<<<params.grid_size, params.block_size>>>(
        bh, model);
    CELER_CUDA_CHECK_ERROR();
}

//---------------------------------------------------------------------------//
} // namespace detail
} // namespace celeritas
