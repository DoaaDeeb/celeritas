#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file SeltzerBerger.cu
//---------------------------------------------------------------------------//
#include "SeltzerBerger.hh"

#include "base/Assert.hh"
#include "base/KernelParamCalculator.cuda.hh"
#include "base/StackAllocator.hh"
#include "random/cuda/RngEngine.hh"
#include "physics/base/ModelInterface.hh"
#include "physics/base/ParticleTrackView.hh"
#include "physics/base/PhysicsTrackView.hh"
#include "physics/material/MaterialTrackView.hh"
#include "SeltzerBergerInteractor.hh"

namespace celeritas
{
namespace detail
{
namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
/*!
 * Interact using the Seltzer-Berger model on applicable tracks.
 */
__global__ void
seltzer_berger_interact_kernel(const SeltzerBergerDeviceRef sb,
                               const ModelInteractPointers  model)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= model.states.size())
        return;

    StackAllocator<Secondary> allocate_secondaries(model.secondaries);
    ParticleTrackView      particle(
        model.params.particle, model.states.particle, tid);

    // Setup for ElementView access
    MaterialTrackView material(
        model.params.material, model.states.material, tid);
    // Cache the associated MaterialView as function calls to MaterialTrackView
    // are expensive
    MaterialView material_view = material.material_view();

    PhysicsTrackView physics(model.params.physics,
                             model.states.physics,
                             particle.particle_id(),
                             material.material_id(),
                             tid);

    // This interaction only applies if the Seltzer-Berger model was selected
    if (physics.model_id() != sb.ids.model)
        return;

    // Assume only a single element in the material, for now
    CELER_ASSERT(material_view.num_elements() == 1);
    SeltzerBergerInteractor interact(
        sb,
        particle,
        model.states.direction[tid.get()],
        allocate_secondaries,
        material_view.element_view(celeritas::ElementComponentId{0}));

    RngEngine rng(model.states.rng, tid);
    model.result[tid.get()] = interact(rng);
    CELER_ENSURE(model.result[tid.get()]);
}

} // namespace

//---------------------------------------------------------------------------//
// LAUNCHERS
//---------------------------------------------------------------------------//
/*!
 * Launch the Seltzer-Berger interaction kernel.
 */
void seltzer_berger_interact(const SeltzerBergerDeviceRef& sb,
                             const ModelInteractPointers&  model)
{
    CELER_EXPECT(sb);
    CELER_EXPECT(model);

    static const KernelParamCalculator calc_kernel_params(
        seltzer_berger_interact_kernel, "seltzer_berger_interact");
    auto params = calc_kernel_params(model.states.size());
    seltzer_berger_interact_kernel<<<params.grid_size, params.block_size>>>(
        sb, model);
    CELER_CUDA_CHECK_ERROR();
}

//---------------------------------------------------------------------------//
} // namespace detail
} // namespace celeritas
